#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <random>
#include <hipcub/hipcub.hpp>
#define VECTOR_SIZE 4096

void printArray(const char* name, const float* array, int size, int limit = 10) {
	std::cout << name << ": ";
	for (int i = 0; i < std::min(size, limit); i++)
		std::cout << array[i] << " ";
	std::cout << "..." << std::endl;
}

bool areFloatArrayEqual(const float* x, const float* y, int size, float epsilon = 1e-1) {
	for (size_t i = 0; i < size; i++) {
		if (std::fabs(x[i] - y[i]) > epsilon) {
			std::cout << "Mismatch at index " << i << ":" << x[i] << " vs " << y[i] << std::endl;
			return false;
		}
	}
	return true;
}

__global__
void kogge_stone_scan_double_buffer_kernel(float* x_d, float* y_d, int n) {
	extern __shared__ float buffer[];
	float* XY0 = buffer;
	float* XY1 = buffer + blockDim.x;

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	// �����ȼ��ص�������0
	if (i < n) {
		XY0[threadIdx.x] = x_d[i];
	}
	else {
		XY0[threadIdx.x] = 0.0f;
	}
	bool flag = true;
	for (size_t step = 1; step < blockDim.x; step *= 2) {
		__syncthreads();
		if (flag) {
			if (threadIdx.x >= step)
				XY1[threadIdx.x] = XY0[threadIdx.x] + XY0[threadIdx.x - step];
			else
				XY1[threadIdx.x] = XY0[threadIdx.x];
			flag = false;
		}
		else {
			if (threadIdx.x >= step)
				XY0[threadIdx.x] = XY1[threadIdx.x] + XY1[threadIdx.x - step];
			else
				XY0[threadIdx.x] = XY1[threadIdx.x];
			flag = true;
		}
	}
	__syncthreads();
	if (i < n) {
		if(flag)
			y_d[i] = XY0[threadIdx.x];
		else
			y_d[i] = XY1[threadIdx.x];
	}
}
void kogge_stone_scan_double_buffer(float* x_h, float* y_h, int n) {
	float* x_d, * y_d;
	hipMalloc((void**)&x_d, n * sizeof(float));
	hipMalloc((void**)&y_d, n * sizeof(float));
	hipMemcpy(x_d, x_h, n * sizeof(float), hipMemcpyHostToDevice);
	dim3 blockSize(n);
	dim3 blockNum((n + blockSize.x - 1) / blockSize.x);
	kogge_stone_scan_double_buffer_kernel << <blockNum, blockSize, 2 * blockSize.x * sizeof(float) >> > (x_d, y_d, n);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
	}
	hipMemcpy(y_h, y_d, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(x_d);
	hipFree(y_d);
}
__global__
void kogge_stone_scan_kernel(float* x_d, float* y_d, int n) {
	extern __shared__ float XY[];
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		XY[threadIdx.x] = x_d[i];
	}
	else {
		XY[threadIdx.x] = 0.0f;
	}
	for (size_t step = 1; step < blockDim.x; step *= 2) {
		__syncthreads();
		float temp = 0.0f;
		if (threadIdx.x >= step) {
			temp = XY[threadIdx.x] + XY[threadIdx.x - step];
		}
		__syncthreads();
		if (threadIdx.x >= step) {
			XY[threadIdx.x] = temp;
		}
	}
	if (i < n) {
		y_d[i] = XY[threadIdx.x];
	}
}
void kogge_stone_scan(float* x_h, float* y_h, int n) {
	float* x_d, * y_d;
	hipMalloc((void**)&x_d, n * sizeof(float));
	hipMalloc((void**)&y_d, n * sizeof(float));
	hipMemcpy(x_d, x_h, n * sizeof(float), hipMemcpyHostToDevice);
	dim3 blockSize(n);
	dim3 blockNum((n + blockSize.x - 1) / blockSize.x);
	kogge_stone_scan_kernel << <blockNum, blockSize, blockSize.x * sizeof(float) >> > (x_d, y_d, n);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
	}
	hipMemcpy(y_h, y_d, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(x_d);
	hipFree(y_d);
}
__global__
void brent_kung_scan_kernel(float* x_d, float* y_d, int n) {
	extern __shared__ float XY[];
	size_t i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) 
		XY[threadIdx.x] = x_d[i];
	else
		XY[threadIdx.x] = 0.0f;
	if (i + blockDim.x < n)
		XY[threadIdx.x + blockDim.x] = x_d[i + blockDim.x];
	else
		XY[threadIdx.x + blockDim.x] = 0.0f;
	for (size_t step = 1; step <= blockDim.x; step *= 2) {
		__syncthreads();
		size_t index = (threadIdx.x + 1) * 2 * step - 1;
		if (index < n)
			XY[index] += XY[index - step];
	}
	
	for (size_t step = blockDim.x / 2; step > 0; step /= 2) {
		__syncthreads();
		size_t index = (threadIdx.x + 1) * 2 * step - 1;
		if (index + step < n)
			XY[index + step] += XY[index];
	}
	__syncthreads();
	if (i < n)
		y_d[i] = XY[threadIdx.x];
	if (i + blockDim.x < n)
		y_d[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
}
void brent_kung_scan(float* x_h, float* y_h, int n) {
	float* x_d, * y_d;
	hipMalloc((void**)&x_d, n * sizeof(float));
	hipMalloc((void**)&y_d, n * sizeof(float));
	hipMemcpy(x_d, x_h, n * sizeof(float), hipMemcpyHostToDevice);
	dim3 blockSize(n/2);
	dim3 blockNum((n/2 + blockSize.x - 1) / blockSize.x);
	brent_kung_scan_kernel << <blockNum, blockSize, n * sizeof(float) >> > (x_d, y_d, n);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
	}
	hipMemcpy(y_h, y_d, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(x_d);
	hipFree(y_d);
}
/*Thread coarsening*/
__global__
void three_phase_parallel_inclusive_scan_kernel(float* x_d, float* y_d, int n,int shared_mem_size) {
	extern __shared__ float XY[];
	// ͬһ���߳̿��У�һ���̴߳���������
	int start_pos = blockIdx.x * shared_mem_size;
	int tx = threadIdx.x;
	// ���������ݴ�ȫ���ڴ���˵������ڴ���
	for (int j = tx; j < shared_mem_size; j += blockDim.x) {
		XY[j] = start_pos + j < n ? x_d[start_pos + j] : 0.0f;
	}
	__syncthreads();
	// �׶�һ������˳��ɨ��
	int coarse_size = shared_mem_size / blockDim.x;
	int start = coarse_size * tx;
	int stop = start + coarse_size;
	float temp = 0.0f;
	if (start_pos + start < n) {    // ���һ������޷���ȫ
		for (int i = start; i < stop; i++) {
			temp += XY[i];
			XY[i] = temp;
		}
	}
	__syncthreads();
	// �׶ζ�����䲢��ɨ��  ʹ��Brent-Kung�㷨
	// ��ÿһ������һ��Ԫ�ؽ��в���ɨ��
	for (size_t step = 1; step < blockDim.x; step *= 2) {
		size_t index = (tx + 1) * 2 * step * coarse_size - 1;
		if (index < shared_mem_size)
			XY[index] += XY[index - step * coarse_size];
		__syncthreads();
	}
	for (size_t step = shared_mem_size/4; step >=coarse_size; step /= 2) {
		size_t index = (tx + 1) * 2 * step - 1;
		if (index + step < shared_mem_size)
			XY[index + step] += XY[index];
		__syncthreads();
	}
	// �׶�������ÿһ������һ��Ԫ�ؼӵ���һ���ǰ����Ԫ����
	if (tx != 0) {
		float value = XY[start - 1];
		for (int i = start; i < stop - 1; i++) {
			XY[i] += value;
		}
	}
	__syncthreads();
	//���ؽ����ȫ���ڴ���
	for (int i = tx; i < shared_mem_size; i += blockDim.x) {
		if (start_pos + i < n) {
			y_d[start_pos + i] = XY[i];
		}
	}
}
void three_phase_parallel_inclusive_scan(float* x_h, float* y_h, int n) {
	float* x_d, * y_d;
	hipMalloc((void**)&x_d, n * sizeof(float));
	hipMalloc((void**)&y_d, n * sizeof(float));
	hipMemcpy(x_d, x_h, n * sizeof(float), hipMemcpyHostToDevice);
	dim3 blockSize(1024);
	dim3 blockNum((n / 4 + blockSize.x - 1) / blockSize.x);
	three_phase_parallel_inclusive_scan_kernel << <blockNum.x, blockSize.x, n * sizeof(float) >> > (x_d, y_d,n, n);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
	}
	hipMemcpy(y_h, y_d, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(x_d);
	hipFree(y_d);
}

//cuda_cub
void cub_inclusive_scan(float* x_h, float* y_h, int n) {
	float* x_d, * y_d;
	hipMalloc((void**)&x_d, n * sizeof(float));
	hipMalloc((void**)&y_d, n * sizeof(float));
	hipMemcpy(x_d, x_h, n * sizeof(float), hipMemcpyHostToDevice);

	void* d_temp_storage = nullptr;
	size_t temp_storage_bytes = 0;
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, x_d, y_d, n);
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, x_d, y_d, n);
	hipMemcpy(y_h, y_d, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_temp_storage);
	hipFree(x_d);
	hipFree(y_d);
}

int main() {
	float* x_h = (float*)malloc(VECTOR_SIZE * sizeof(float));
	// �����������
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<float> dist(0.0f, 100.0f);

	for (int i = 0; i < VECTOR_SIZE; i++) {
		x_h[i] = dist(gen);
	}

	float* y_h_cub = (float*)malloc(VECTOR_SIZE * sizeof(float));
	float* y_h_gpu = (float*)malloc(VECTOR_SIZE * sizeof(float));

	three_phase_parallel_inclusive_scan(x_h, y_h_gpu, VECTOR_SIZE);
	cub_inclusive_scan(x_h, y_h_cub, VECTOR_SIZE);
	if (areFloatArrayEqual(y_h_cub, y_h_gpu, VECTOR_SIZE))
		std::cout << "true"<<std::endl;
	else
		std::cout << "false"<<std::endl;
	printArray("src", x_h, VECTOR_SIZE);
	printArray("cub", y_h_cub, VECTOR_SIZE);
	printArray("gpu", y_h_gpu, VECTOR_SIZE);
	free(x_h);
	free(y_h_cub);
	free(y_h_gpu);
	return 0;
}